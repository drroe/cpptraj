#include "hip/hip_runtime.h"
#include "kWaterShell.cuh"
#include "NonOrtho_dist2.cuh"
#include "ortho_dist2.cuh"
#include <cstdio> // DEBUG

// -----------------------------------------------------------------------------
/** Calculate # waters in 1st and 2nd solvation shells based on distance cutoffs. */
__global__ void kWaterShell_NoImage(const CpptrajGpu::FpType* xyz1, int N1, const CpptrajGpu::FpType* xyz2, int N2,
                                    CpptrajGpu::FpType lowerCut2, CpptrajGpu::FpType upperCut2, int* VatomShell)
{
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;

  if (a1 < N1 && a2 < N2) {
    int idx1 = a1 * 3;
    CpptrajGpu::FpType a1x = xyz1[idx1  ];
    CpptrajGpu::FpType a1y = xyz1[idx1+1];
    CpptrajGpu::FpType a1z = xyz1[idx1+2];

    int idx2 = a2 * 3;
    CpptrajGpu::FpType x = a1x - xyz2[idx2  ];
    CpptrajGpu::FpType y = a1y - xyz2[idx2+1];
    CpptrajGpu::FpType z = a1z - xyz2[idx2+2];

    CpptrajGpu::FpType dist2 = (x*x) + (y*y) + (z*z);
    if (dist2 < upperCut2) {
      VatomShell[a1] = 2;
      if (dist2 < lowerCut2) {
        printf("a1= %i  a2= %i  dist= %f lower.\n", a1+1, a2+1, sqrt(dist2));
        VatomShell[a1] = 1;
      }
    }
  }
}

/** Calculate # waters in 1st and 2nd solvation shells based on distance cutoffs. */
__global__ void kWaterShell_Ortho(const CpptrajGpu::FpType* xyz1, int N1, const CpptrajGpu::FpType* xyz2, int N2,
                                  const CpptrajGpu::FpType* box,
                                  CpptrajGpu::FpType lowerCut2, CpptrajGpu::FpType upperCut2, int* VatomShell)
{
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;

  if (a1 < N1 && a2 < N2) {
    int idx1 = a1 * 3;
    CpptrajGpu::FpType a1x = xyz1[idx1  ];
    CpptrajGpu::FpType a1y = xyz1[idx1+1];
    CpptrajGpu::FpType a1z = xyz1[idx1+2];

    int idx2 = a2 * 3;
    CpptrajGpu::FpType a2x = xyz2[idx2  ];
    CpptrajGpu::FpType a2y = xyz2[idx2+1];
    CpptrajGpu::FpType a2z = xyz2[idx2+2];

    CpptrajGpu::FpType dist2 = ortho_dist2<CpptrajGpu::FpType>(a1x, a1y, a1z, a2x, a2y, a2z, box);
    if (dist2 < upperCut2) {
      VatomShell[a1] = 2;
      if (dist2 < lowerCut2)
        VatomShell[a1] = 1;
    }
  }
}

/** Calculate # waters in 1st and 2nd solvation shells based on distance cutoffs. */
__global__ void kWaterShell_nonOrtho(const CpptrajGpu::FpType* xyz1, int N1, const CpptrajGpu::FpType* xyz2, int N2,
                                     const CpptrajGpu::FpType* frac, const CpptrajGpu::FpType* ucell,
                                     CpptrajGpu::FpType lowerCut2, CpptrajGpu::FpType upperCut2, int* VatomShell)
{
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;

  if (a1 < N1 && a2 < N2) {
    int idx1 = a1 * 3;
    CpptrajGpu::FpType a1x = xyz1[idx1  ];
    CpptrajGpu::FpType a1y = xyz1[idx1+1];
    CpptrajGpu::FpType a1z = xyz1[idx1+2];
    CpptrajGpu::FpType f1x = frac[0]*a1x + frac[1]*a1y + frac[2]*a1z;
    CpptrajGpu::FpType f1y = frac[3]*a1x + frac[4]*a1y + frac[5]*a1z;
    CpptrajGpu::FpType f1z = frac[6]*a1x + frac[7]*a1y + frac[8]*a1z;

    int idx2 = a2 * 3;
    CpptrajGpu::FpType a2x = xyz2[idx2  ];
    CpptrajGpu::FpType a2y = xyz2[idx2+1];
    CpptrajGpu::FpType a2z = xyz2[idx2+2];
    CpptrajGpu::FpType f2x = frac[0]*a2x + frac[1]*a2y + frac[2]*a2z;
    CpptrajGpu::FpType f2y = frac[3]*a2x + frac[4]*a2y + frac[5]*a2z;
    CpptrajGpu::FpType f2z = frac[6]*a2x + frac[7]*a2y + frac[8]*a2z;

    CpptrajGpu::FpType dist2 =  NonOrtho_dist2<CpptrajGpu::FpType>(f2x, f2y, f2z, f1x ,f1y, f1z, ucell);
    if (dist2 < upperCut2) {
      VatomShell[a1] = 2;
      if (dist2 < lowerCut2)
        VatomShell[a1] = 1;
    }
  }
}
