#include "wrapper_WaterShell.cuh"
#include "kWaterShell.cuh"
#include "cuda_box.cuh"
#include "../CpptrajStdio.h"
#if defined(__HIP_PLATFORM_HCC__)
#include <hip/hip_runtime.h>
#include "../HipDefinitions.h"
#endif

static inline int calc_nblocks(int ntotal, int nthreadsPerBlock)
{
  int nblocks = ntotal / nthreadsPerBlock;
  if ( (ntotal % nthreadsPerBlock) != 0 )
    nblocks++;
  return nblocks;
}

/** Report any cuda errors. */
static inline int Cuda_check(hipError_t err, const char* desc) {
  //hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    mprintf("Warning: CUDA Error %s: %s\n", desc, hipGetErrorString(err));
    mprinterr("Error: CUDA Error %s: %s\n", desc, hipGetErrorString(err));
    //return 1;
  }
  return 0;
}

/** Calculate distances between pairs of atoms and bin them into a 1D histogram. */
int Cpptraj_GPU_WaterShell(int* VatomShell,
                           CpptrajGpu::FpType lowerCut2, CpptrajGpu::FpType upperCut2,
                           const CpptrajGpu::FpType* xyz1, int N1,
                           const CpptrajGpu::FpType* xyz2, int N2,
                           ImageOption::Type imageType,
                           CpptrajGpu::HostBox<CpptrajGpu::FpType> const& box)
{
  int* device_vatomshell;
  Cuda_check(hipMalloc(((void**)(&device_vatomshell)), N1 * sizeof(int)), "Allocating solvent atom shell array");
  Cuda_check(hipMemset( device_vatomshell, 0, N1*sizeof(int) ), "Setting solvent atom shells to 0");

  CpptrajGpu::FpType* device_xyz1;
  Cuda_check(hipMalloc(((void**)(&device_xyz1)), N1 * 3 * sizeof(CpptrajGpu::FpType)), "Allocating xyz1");
  Cuda_check(hipMemcpy(device_xyz1, xyz1, N1 * 3 * sizeof(CpptrajGpu::FpType), hipMemcpyHostToDevice), "Copying xyz1");

  CpptrajGpu::FpType* device_xyz2;
  Cuda_check(hipMalloc(((void**)(&device_xyz2)), N2 * 3 * sizeof(CpptrajGpu::FpType)), "Allocating xyz2");
  Cuda_check(hipMemcpy(device_xyz2, xyz2, N2 * 3 * sizeof(CpptrajGpu::FpType), hipMemcpyHostToDevice), "Copying xyz2");

  cuda_box<CpptrajGpu::FpType> gpuBox;
  if ( gpuBox.Setup( imageType, box.BoxLengths(), box.Ucell(), box.Frac() ) ) {
    Cuda_check( gpuBox.LastErr(), gpuBox.LastErrDesc() );
    return 1;
  }

  // Determine number of blocks
  unsigned int BLOCKDIM = CpptrajGpu::MaxBlockDim_2D();

  dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
  dim3 numBlocks(calc_nblocks(N1, threadsPerBlock.x), calc_nblocks(N2, threadsPerBlock.y));
  mprintf("#Atoms = %i, %i; Threads per block = %i, %i;  #Blocks = %i, %i\n",
          N1, N2, threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);

  // Launch kernel
  // Must have Non-overlapping coords
  switch (imageType) {
      case ImageOption::NONORTHO:
        kWaterShell_nonOrtho<<<numBlocks, threadsPerBlock>>>(
          device_xyz1, N1, device_xyz2, N2, gpuBox.FracDev(), gpuBox.UcellDev(), lowerCut2, upperCut2, device_vatomshell);
      break;
      case ImageOption::ORTHO:
        kWaterShell_Ortho<<<numBlocks, threadsPerBlock>>>(
          device_xyz1, N1, device_xyz2, N2, gpuBox.BoxDev(), lowerCut2, upperCut2, device_vatomshell);
      break;
      case ImageOption::NO_IMAGE:
        kWaterShell_NoImage<<<numBlocks, threadsPerBlock>>>(
          device_xyz1, N1, device_xyz2, N2, lowerCut2, upperCut2, device_vatomshell);
      break;
      //default:
      //  mprinterr("Internal Error: kernel_rdf: Unhandled image type.\n");
      //  return 1;
  }

  // Error check
  Cuda_check(hipGetLastError(), "watershell kernel launch");

  // Copy the result back
  //int* local_counts[2];
  hipMemcpy(VatomShell, device_vatomshell, N1*sizeof(int), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(device_vatomshell);
  hipFree(device_xyz1);
  hipFree(device_xyz2);

  return 0;
}
