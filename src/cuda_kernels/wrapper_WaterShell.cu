#include "wrapper_WaterShell.cuh"
#include "kWaterShell.cuh"
#include "cuda_box.cuh"
#include "../CpptrajStdio.h"
#if defined(__HIP_PLATFORM_HCC__)
#include <hip/hip_runtime.h>
#include "../HipDefinitions.h"
#endif

static inline int calc_nblocks(int ntotal, int nthreadsPerBlock)
{
  int nblocks = ntotal / nthreadsPerBlock;
  if ( (ntotal % nthreadsPerBlock) != 0 )
    nblocks++;
  return nblocks;
}

/** Report any cuda errors. */
static inline int Cuda_check(hipError_t err, const char* desc) {
  //hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    mprintf("Warning: CUDA Error %s: %s\n", desc, hipGetErrorString(err));
    mprinterr("Error: CUDA Error %s: %s\n", desc, hipGetErrorString(err));
    //return 1;
  }
  return 0;
}

/** Calculate distances between pairs of atoms and bin them into a 1D histogram. */
int Cpptraj_GPU_WaterShell(int& nlower, int& nupper,
                           CpptrajGpu::FpType lowerCut2, CpptrajGpu::FpType upperCut2,
                           const CpptrajGpu::FpType* xyz1, int N1,
                           const CpptrajGpu::FpType* xyz2, int N2,
                           ImageOption::Type imageType,
                           CpptrajGpu::HostBox<CpptrajGpu::FpType> const& box)
{
  int* device_counts;
  Cuda_check(hipMalloc(((void**)(&device_counts)), 2 * sizeof(int)), "Allocating watershell bins");
  Cuda_check(hipMemset( device_counts, 0, 2*sizeof(int) ), "Setting watershell bins to 0");

  CpptrajGpu::FpType* device_xyz1;
  Cuda_check(hipMalloc(((void**)(&device_xyz1)), N1 * 3 * sizeof(CpptrajGpu::FpType)), "Allocating xyz1");
  Cuda_check(hipMemcpy(device_xyz1, xyz1, N1 * 3 * sizeof(CpptrajGpu::FpType), hipMemcpyHostToDevice), "Copying xyz1");

  CpptrajGpu::FpType* device_xyz2;
  Cuda_check(hipMalloc(((void**)(&device_xyz2)), N2 * 3 * sizeof(CpptrajGpu::FpType)), "Allocating xyz2");
  Cuda_check(hipMemcpy(device_xyz2, xyz2, N2 * 3 * sizeof(CpptrajGpu::FpType), hipMemcpyHostToDevice), "Copying xyz2");

  cuda_box<CpptrajGpu::FpType> gpuBox;
  if ( gpuBox.Setup( imageType, box.BoxLengths(), box.Ucell(), box.Frac() ) ) {
    Cuda_check( gpuBox.LastErr(), gpuBox.LastErrDesc() );
    return 1;
  }

  // Determine number of blocks
  unsigned int BLOCKDIM = CpptrajGpu::MaxBlockDim_2D();

  dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
  dim3 numBlocks(calc_nblocks(N1, threadsPerBlock.x), calc_nblocks(N2, threadsPerBlock.y));
  mprintf("#Atoms = %i, %i; Threads per block = %i, %i;  #Blocks = %i, %i\n",
          N1, N2, threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);

  // Launch kernel
  // Must have Non-overlapping coords
  mprintf("DEBUG: before launch: nlower= %i nupper= %i\n", nlower, nupper);
  switch (imageType) {
      case ImageOption::NONORTHO:
        kWaterShell_nonOrtho<<<numBlocks, threadsPerBlock>>>(
          device_xyz1, N1, device_xyz2, N2, gpuBox.FracDev(), gpuBox.UcellDev(), lowerCut2, upperCut2, device_counts);
      break;
      case ImageOption::ORTHO:
        kWaterShell_Ortho<<<numBlocks, threadsPerBlock>>>(
          device_xyz1, N1, device_xyz2, N2, gpuBox.BoxDev(), lowerCut2, upperCut2, device_counts);
      break;
      case ImageOption::NO_IMAGE:
        kWaterShell_NoImage<<<numBlocks, threadsPerBlock>>>(
          device_xyz1, N1, device_xyz2, N2, lowerCut2, upperCut2, device_counts);
      break;
      //default:
      //  mprinterr("Internal Error: kernel_rdf: Unhandled image type.\n");
      //  return 1;
  }
  mprintf("DEBUG: after launch: nlower= %i nupper= %i\n", nlower, nupper);

  // Error check
  Cuda_check(hipGetLastError(), "watershell kernel launch");

  // Copy the result back
  int local_counts[2];
  hipMemcpy(local_counts, device_counts, 2*sizeof(int), hipMemcpyDeviceToHost);
  nlower = local_counts[0];
  nupper = local_counts[1];

  // Free device memory
  hipFree(device_counts);
  hipFree(device_xyz1);
  hipFree(device_xyz2);

  return 0;
}
