#include "hip/hip_runtime.h"
#include "kRDF.cuh"
#include "NonOrtho_dist2.cuh"
#include "ortho_dist2.cuh"
//#include <cstdio> // DEBUG

// -----------------------------------------------------------------------------
/** Bin distances from two non-overlapping sets of coords. */
__global__ void kBinDistances_nonOverlap_NoImage(int* RDF,
                                               const CpptrajGpu::FpType* xyz1, int N1, const CpptrajGpu::FpType* xyz2, int N2,
                                               CpptrajGpu::FpType maximum2, CpptrajGpu::FpType one_over_spacing)
{
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;

  if (a1 < N1 && a2 < N2) {
    int idx1 = a1 * 3;
    CpptrajGpu::FpType a1x = xyz1[idx1  ];
    CpptrajGpu::FpType a1y = xyz1[idx1+1];
    CpptrajGpu::FpType a1z = xyz1[idx1+2];

    int idx2 = a2 * 3;
    CpptrajGpu::FpType x = a1x - xyz2[idx2  ];
    CpptrajGpu::FpType y = a1y - xyz2[idx2+1];
    CpptrajGpu::FpType z = a1z - xyz2[idx2+2];

    CpptrajGpu::FpType dist2 = (x*x) + (y*y) + (z*z); 
    if (dist2 > 0 && dist2 <= maximum2) {
      CpptrajGpu::FpType dist = sqrt(dist2);
      int histIdx = (int) (dist * one_over_spacing);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i\n", a1+1, a2+1, dist, histIdx);
      //printf("DEBUG: xyz1= %f %f %f\n", a1x, a1y, a1z);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i  xyz1=%f %f %f  xyz2=%f %f %f\n", a1+1, a2+1, dist, histIdx,
      //       a1x, a1y, a1z, a2x, a2y, a2z);
      atomicAdd( RDF + histIdx, 1 );
    }
  }
}

/** Bin distances from two non-overlapping sets of coords. */
__global__ void kBinDistances_nonOverlap_Ortho(int* RDF,
                                               const CpptrajGpu::FpType* xyz1, int N1, const CpptrajGpu::FpType* xyz2, int N2,
                                               const CpptrajGpu::FpType* box,
                                               CpptrajGpu::FpType maximum2, CpptrajGpu::FpType one_over_spacing)
{
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;

  if (a1 < N1 && a2 < N2) {
    int idx1 = a1 * 3;
    CpptrajGpu::FpType a1x = xyz1[idx1  ];
    CpptrajGpu::FpType a1y = xyz1[idx1+1];
    CpptrajGpu::FpType a1z = xyz1[idx1+2];

    int idx2 = a2 * 3;
    CpptrajGpu::FpType a2x = xyz2[idx2  ];
    CpptrajGpu::FpType a2y = xyz2[idx2+1];
    CpptrajGpu::FpType a2z = xyz2[idx2+2];

    CpptrajGpu::FpType dist2 = ortho_dist2<CpptrajGpu::FpType>(a1x, a1y, a1z, a2x, a2y, a2z, box);
    if (dist2 > 0 && dist2 <= maximum2) {
      CpptrajGpu::FpType dist = sqrt(dist2);
      int histIdx = (int) (dist * one_over_spacing);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i\n", a1+1, a2+1, dist, histIdx);
      //printf("DEBUG: xyz1= %f %f %f\n", a1x, a1y, a1z);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i  xyz1=%f %f %f  xyz2=%f %f %f\n", a1+1, a2+1, dist, histIdx,
      //       a1x, a1y, a1z, a2x, a2y, a2z);
      atomicAdd( RDF + histIdx, 1 );
    }
  }
}

/** Bin distances from two non-overlapping sets of coords. */
__global__ void kBinDistances_nonOverlap_nonOrtho(int* RDF,
                                                  const CpptrajGpu::FpType* xyz1, int N1, const CpptrajGpu::FpType* xyz2, int N2,
                                                  const CpptrajGpu::FpType* frac, const CpptrajGpu::FpType* ucell,
                                                  CpptrajGpu::FpType maximum2, CpptrajGpu::FpType one_over_spacing)
{
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;

  if (a1 < N1 && a2 < N2) {
    int idx1 = a1 * 3;
    CpptrajGpu::FpType a1x = xyz1[idx1  ];
    CpptrajGpu::FpType a1y = xyz1[idx1+1];
    CpptrajGpu::FpType a1z = xyz1[idx1+2];
    CpptrajGpu::FpType f1x = frac[0]*a1x + frac[1]*a1y + frac[2]*a1z;
    CpptrajGpu::FpType f1y = frac[3]*a1x + frac[4]*a1y + frac[5]*a1z;
    CpptrajGpu::FpType f1z = frac[6]*a1x + frac[7]*a1y + frac[8]*a1z;

    int idx2 = a2 * 3;
    CpptrajGpu::FpType a2x = xyz2[idx2  ];
    CpptrajGpu::FpType a2y = xyz2[idx2+1];
    CpptrajGpu::FpType a2z = xyz2[idx2+2];
    CpptrajGpu::FpType f2x = frac[0]*a2x + frac[1]*a2y + frac[2]*a2z;
    CpptrajGpu::FpType f2y = frac[3]*a2x + frac[4]*a2y + frac[5]*a2z;
    CpptrajGpu::FpType f2z = frac[6]*a2x + frac[7]*a2y + frac[8]*a2z;

    CpptrajGpu::FpType dist2 =  NonOrtho_dist2<CpptrajGpu::FpType>(f2x, f2y, f2z, f1x ,f1y, f1z, ucell);
    if (dist2 > 0 && dist2 <= maximum2) {
      CpptrajGpu::FpType dist = sqrt(dist2);
      int histIdx = (int) (dist * one_over_spacing);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i\n", a1+1, a2+1, dist, histIdx);
      //printf("DEBUG: xyz1= %f %f %f\n", a1x, a1y, a1z);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i  xyz1=%f %f %f  xyz2=%f %f %f\n", a1+1, a2+1, dist, histIdx,
      //       a1x, a1y, a1z, a2x, a2y, a2z);
      atomicAdd( RDF + histIdx, 1 );
    }
  }
}
